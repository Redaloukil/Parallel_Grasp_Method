#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
// vector::push_back
#include<iostream>
#include"parcel.cpp"


#include <thrust/device_vector.h>

#include<algorithm>

__GLOBAL__
void SelectionPhase(vector<Parcel> *parcels ,Rcl *rcl  ,Cordinate position){
    
    //thread block dimentions 
    int t = threadIdx.x;
    int T = blockDim.x;

    //malloc a list local restricted candidates list
    thrust::device_vector<SelectParcel> LocalRcl(4);

    //map function : for each evaluate cost and push to local rcl
    for (int i = t ; i < parcels.size() ; i += T){
        float cost = calculateCost(position , parcels->parcels[i]);
        if(cost <= item){
            int index = i;
            SelectParcel parcel ; parcel.index = index;
            parcel.parcel = parcels->parcels[i];
            printf("selected\n");
            LocalRcl.push_back(parcel);
        }else {
            printf("not selected\n");
        }
    }
    //select randomly from rcl
    int elem = randBetweenInt(0 , rcl.size());
    Parcel selectedParcel = rcl[elem];
    
    //push selected parcel to the global rcl 
    rcl.push_back(selectedParcel) = selectedParcel;
    
}  