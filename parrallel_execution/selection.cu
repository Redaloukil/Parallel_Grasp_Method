#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
// vector::push_back
#include<iostream>
#include"parcel.cpp"
#include<vector>
#include<algorithm>

__GLOBAL__ 
void SelectionPhase(vector<Parcel> *parcels ,Rcl *rcl  ,Cordinate position){
    //malloc a list local restricted candidates list
    Rcl localRcl;
    
    //thread block dimentions 
    int t = threadIdx.x;
    int T = blockDim.x;

    //map function : for each evaluate cost and push to local rcl
    for (int i = t;i<parcels->parcels.size(); i += T){
        float cost = calculateCost(position , parcels->parcels[i]);
        if(cost <= item){
            int index = i;
            SelectParcel parcel;parcel.index = index;
            parcel.parcel = parcels->parcels[i];
            printf("selected\n");
            localRcl->parcels.push_back(parcel);
        }else {
            printf("not selected\n");
        }
    }
    //select randomly from rcl
    int elem = randBetweenInt(0 , rcl->parcels.size());
    Parcel selectedParcel = rcl->parcels[elem];
    
    //push selected parcel to the global rcl 
    rcl->parcels[t] = selectedParcel;
    
    
}  